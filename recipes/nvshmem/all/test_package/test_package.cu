#include <hip/hip_runtime.h>
#include <nvshmem.h>

void dummy() {
    hipStream_t stream;
    nvshmem_init();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);
    nvshmemx_barrier_all_on_stream(stream);
    hipStreamSynchronize(stream);
    nvshmem_finalize();
}

int main() { }
